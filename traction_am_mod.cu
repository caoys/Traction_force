#include "hip/hip_runtime.h"
/*The basic model of Danying's 2012 PNAS paper, without explicitly track the adhesion sites. Instead, using a rhoM dependent manner to handle the friction and then the traction force*/

//Feb-22-2018: Looks like for the diffusion term, spectral method does not work well. This is the only term that used finite differetiation in this code
// may-23-2020: set the rhoa at front half and rhom at back half
// July-26-2020: traction force for amoeboid cells, the protrusions are modeled by a random fiber
// Mar-13-2021: amoeboid cells wave patterns updated

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include "hipblas.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>

#include <hipfft/hipfft.h>

#include <stdio.h>
#include<stdlib.h>
#include"math.h"
#include<algorithm>
#include<random>
#include<time.h>
#include <chrono>

#define MIN(a,b) ((a<b) ? a:b)
#define MAX(a,b) ((a>b) ? a:b)


# define M_PI           3.14159265358979323846  /* pi */
#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16
using namespace std;

//simulation grid set up
const int Nx = 256;
const int Ny = 256;
dim3 blocks(Nx / BLOCK_SIZE_X, Ny / BLOCK_SIZE_Y);
dim3 threadsperblock(BLOCK_SIZE_X, BLOCK_SIZE_Y);


//grid box
const float Lx = 25.0f, Ly = 25.0f;
const float dx = 2.0f * Lx / (float)Nx, dy = 2.0f * Ly / (float)Ny;
//time steps
float dt = 2e-3;
//relaxation Gamma
extern __constant__ float d_Gamma = 0.4f;
//phase-field width
extern __constant__ float d_epsilon = 2.0f;
const float h_epsilon = 2.0f;

//runing time and recording time
float max_time = 400.0f;
const float record_itvl = 5.0f;
//intial condition
float rhoAinitial = 1.0f;
float rhoMinitial = 0.3f;

//error control
float poisson_err_limit = 0.01f;
int poisson_max_steps = 100;
float exceed_val = 2.0f; //the exceed value in constructing poisson eq
extern __constant__ float d_exceed_val  = 2.0f;
//const float lamda = 1e-4;

//some global parameters
float r0_cell = 10.0f; //radius of cell
extern __constant__ float d_tension = 20.0f; //tension per unit area
extern __constant__ float d_bending = 0.0f; //bending energy
extern __constant__ float d_Mv = 200.0f; //volume conservation strength
extern __constant__ float d_areaMin = 200.0f; //min-area size 
extern __constant__ float d_areaMax = 600.0f; // max-area size 
extern __constant__ float d_basal_m = 0.0f; //the basal level of myosin at cell boundary
float nu_vis = 1000.0f;

//friction and traction parameters
float xi_fric = 0.5f;
extern __constant__ float d_xiM = 0.0f;
extern __constant__ float d_expRhoM = 0.1f;
extern __constant__ float d_thresA = 0.5f;

float h_diffRhoA = 0.8f;
float h_diffRhoM0 = 2.0f;
float h_KdRhoM = 0.5f;
float h_RhoAtot = 350.0f;
float h_kbRhoA = 10.0f;
float h_kaRhoA = 0.01f;
float h_kcRhoA = 10.0f;
float h_K2aRhoA = 1.0f;

float h_etaRhoA = 2000.0f;
float h_etaRhoM = 80.0f;
float h_period = 80.0f;
float var_p = 40.0f;
float f_radius = 2.0f;
float h_disc_r = 4.0f;
//float A_peri = 100.0f; 

//cufft and cublas handles
hipfftHandle plan_R2C;
hipfftHandle plan_C2R;
hipblasHandle_t blashandle;
hipblasHandle_t h_blashandle;

//utility functions
void Initialize(float* phi, float *rhog, float *rhor, float* ux, float* uy, float* x, float* y);
void writetofile(const char *name, float *u, int Nx, int Ny);
void fftcoeffs(float *d_d1x, float *d_d1y, float *d_d2x, float *d_d2y);
__global__ void absarray(float *absay, float *ax, float *ay, int Nx, int Ny);
__global__ void add3matrix(float *output, float *input1, float *input2, float *input3);
__global__ void add2matrix(float *output, float *input1, float *input2);
__global__ void minus_matrix(float *output, float *minuend, float *substractor, int Nx, int Ny);
__global__ void get_error(float *max_error, float *ux_new, float *ux_old, float *uy_new, float *uy_old, int idx_eux, int idx_ux, int idx_euy, int idx_uy, int Nx, int Ny);
__global__ void matrix_product(float *output, float *input1, float *input2, float alpha, int Nx, int Ny);
__global__ void xdir_center(float *xc_sin, float *xc_cos, float *phi, float *phi_area, float *x, float Lx, int m, int n);
__global__ void ydir_center(float *xc_sin, float *xc_cos, float *phi, float *phi_area, float *x, float Lx, int m, int n);
__global__ void xdir_center(float *xc_sin, float *xc_cos, float *phi, float *phi_area, float *x, float Lx, int label, int m, int n);
__global__ void cylinder_phase(float *output, float start_x, float start_y, float end_x, float end_y, float radius, float *x, float *y, int Nx, int Ny);
__global__ void disc_phase(float *output, int center_idx, float radius, float *x, float *y, int Nx, int Ny);

//derivative functions
void par_deriv1_fft(float *output, float *input, int dim, float *deriv_vec, hipfftComplex *R2C_buffer, hipfftComplex *C2R_buffer, int Nx, int Ny);
void grad_fft(float *output_x, float *output_y, float *input, float *deriv_vec_x, float *deriv_vec_y, hipfftComplex *R2C_buffer, hipfftComplex *C2R_buffer, int Nx, int Ny);
void lap_fft(float *output, float *input, float *deriv_vec_x, float *deriv_vec_y, hipfftComplex *R2C_buffer, hipfftComplex *C2R_buffer, int Nx, int Ny);
void div_fft(float *output, float *input_x, float *input_y, float *deriv_vec_x, float *deriv_vec_y, hipfftComplex *R2C_buffer, hipfftComplex *C2R_buffer, int Nx, int Ny);
__global__ void fftR2C_deriv1(hipfftComplex *input, hipfftComplex *output, int dim, float *coeff, int Nx, int Ny);
__global__ void fftR2C_lap(hipfftComplex *input, hipfftComplex *output, float *coeff_x, float *coeff_y, int Nx, int Ny);
__global__ void poissonR2C(hipfftComplex *input, hipfftComplex *output, float *coeff_x, float *coeff_y, float coeff0, float coeff2, int Nx, int Ny);
void poisson_sol(float *sol, float *rhs, float coeff0, float coeff2, float *coeff_x, float *coeff_y, hipfftComplex *R2C_buffer, hipfftComplex *C2R_buffer, int Nx, int Ny);
void div_pst_fft(float *output, float *c_pst, float *rho, float coeff_diff, float *deriv_vec_x2, float *deriv_vec_y2, float *buffer_x, float *buffer_y, hipfftComplex *R2C_buffer, hipfftComplex *C2R_buffer, int Nx, int Ny);
__global__ void div_pst_fd(float *output, float *c_pst, float *rho, float coeff_diff, float dx, float dy, int Nx, int Ny);

//physical functions
__global__ void curv(float *curv, float *phi_gradx, float *phi_grady, float *absdphi, int Nx, int Ny, float dx, float dy);
__global__ void curv(float *curv, float *phi, float *absdphi, int m, int n, float dx, float dy);
__global__ void phievolve(float *phi_new, float *phi_old, float *grad_x, float *grad_y, float *absdphi, float *lap_phi, float *curv, float *ux, float *uy, int Nx, int Ny, float dt);
__global__ void potential_force(float *output, float *phi, float *lap_phi, float *bendcore, float *phiarea, float A0, float dx, float dy, int Nx, int Ny);
__global__ void rhs_poisson(float *rhs_x, float *rhs_y, float *ptl_force, float *phi, float *dphix, float *dphiy, float *vis_x, float *vis_y, float *act_x, float *act_y, float *rhoA, float *rhoM, float *ux, float *uy, int Nx, int Ny);
__global__ void div_vel_pre(float *div_x, float *div_y, int dim, float *phi, float nu, float *duxdx, float *duxdy, float *duydx, float *duydy, int Nx, int Ny);
__global__ void bend_core(float *core, float *lap_phi, float *phi, int Nx, int Ny);
__global__ void div_advect_pre(float *div_x, float *div_y, float *dnsty, float *rho, float *ux, float *uy, int Nx, int Ny);
__global__ void diffRhoM(float *output, float *phi, float *rhoa, float Dm0, float Kd, int Nx, int Ny);
__global__ void reactionRhoA(float *output, float *phi, float *rhoA, float *rhoAarea, float *phiarea, float rhoAtot, float kb, float K2a, float ka, float kc, float dx, float dy, int Nx, int Ny);
__global__ void RD_evolve(float *rho_new, float *dnsty_new, float *dnsty_old, float *rho_old, float *advect, float *diffuse, float *react, float dt, float lamda, int Nx, int Ny);
__global__ void RD_evolve(float *rho_new, float *dnsty_new, float *dnsty_old, float *rho_old, float *advect, float *diffuse, float dt, float lamda, int Nx, int Ny);
__global__ void normRhoM(float *rho_new, float *rho_area_new, float *rho_are_old, int Nx, int Ny);
__global__ void rhoa_right(float *rhoa, float *phi, float x_sin, float x_cos, float *x, float Lx, int m, int n);
__global__ void activeForce(float *div_x, float *div_y, int dir, float *phi, float etaRhoA, float etaRhoM, float *rhoA, float *rhoM, float *dphix, float *dphiy, int Nx, int Ny);
__global__ void rhoa_am(float *rhoa, float *rhom, float *phi, float *fiber1, float *fiber2, float *fiber3, float t, float period1, float period2, int m, int n);
__global__ void rhoa_osci(float *rhoa, float *rhom, float *phi, float x_sin, float x_cos, float t, float rhoa_period, float *x, float Lx, int m, int n);
__global__ void Gphi(float *output, float *phi, int Nx, int Ny);
__global__ void set_unit(float *output, float alpha, int Nx, int Ny);

int main()
{
	//initialize host
	FILE *ft = fopen("center_traj.txt", "w+");
	float *h_x = (float *)malloc(Nx*sizeof(float));
	float *h_y = (float *)malloc(Ny*sizeof(float));
	float *h_phi = (float *)malloc(Nx*Ny*sizeof(float));
	float *h_ux = (float *)malloc(Nx*Ny*sizeof(float));
	float *h_uy = (float *)malloc(Nx*Ny*sizeof(float));
	float *h_RhoA = (float *)malloc(Nx*Ny*sizeof(float));
	float *h_RhoM = (float *)malloc(Nx*Ny*sizeof(float));
	float h_xsin = 0.0f;
	float h_xcos = 0.0f;
	float x_moc, y_moc;
	float x_f_end, y_f_end;
	float ran_per_1 = h_period;
	float ran_per_2 = h_period;
	float ran_per_3 = h_period / 2.0f;
	int file_record = 1;

	//initialize device
	float *d_x; hipMalloc((void **)&d_x, Nx*sizeof(float));
	float *d_y; hipMalloc((void **)&d_y, Ny*sizeof(float));
	float *d_phi_old; hipMalloc((void **)&d_phi_old, Nx*Ny*sizeof(float));
	float *d_phi_new; hipMalloc((void **)&d_phi_new, Nx*Ny*sizeof(float));
	float *d_ux_old; hipMalloc((void **)&d_ux_old, Nx*Ny*sizeof(float));
	float *d_uy_old; hipMalloc((void **)&d_uy_old, Nx*Ny*sizeof(float));
	float *d_ux_new; hipMalloc((void **)&d_ux_new, Nx*Ny*sizeof(float));
	float *d_uy_new; hipMalloc((void **)&d_uy_new, Nx*Ny*sizeof(float));
	float *d_RhoAold; hipMalloc((void **)&d_RhoAold, Nx*Ny*sizeof(float));
	float *d_RhoAnew; hipMalloc((void **)&d_RhoAnew, Nx*Ny*sizeof(float));
	float *d_RhoMold; hipMalloc((void **)&d_RhoMold, Nx*Ny*sizeof(float));
	float *d_RhoMnew; hipMalloc((void **)&d_RhoMnew, Nx*Ny*sizeof(float));
	float *d_fiber_1; hipMalloc((void **)&d_fiber_1, Nx*Ny*sizeof(float));
	float *d_fiber_2; hipMalloc((void **)&d_fiber_2, Nx*Ny*sizeof(float));
	float *d_fiber_3; hipMalloc((void **)&d_fiber_3, Nx*Ny*sizeof(float));
	float *d_disc_1; hipMalloc((void **)&d_disc_1, Nx*Ny*sizeof(float));
	float *d_disc_2; hipMalloc((void **)&d_disc_2, Nx*Ny*sizeof(float));
	float *d_disc_3; hipMalloc((void **)&d_disc_3, Nx*Ny*sizeof(float));

	//derivatives of velocities
	float *d_duxdx; hipMalloc((void **)&d_duxdx, Nx*Ny*sizeof(float));
	float *d_duxdy; hipMalloc((void **)&d_duxdy, Nx*Ny*sizeof(float));
    	
	float *d_duydx; hipMalloc((void **)&d_duydx, Nx*Ny*sizeof(float));
	float *d_duydy; hipMalloc((void **)&d_duydy, Nx*Ny*sizeof(float));
	//divergence of the viscosity tensor
	float *d_div_x; hipMalloc((void **)&d_div_x, Nx*Ny*sizeof(float));
	float *d_div_y; hipMalloc((void **)&d_div_y, Nx*Ny*sizeof(float));
	//rhs of the poisson equation
	float *d_rhs_ux; hipMalloc((void **)&d_rhs_ux, Nx*Ny*sizeof(float));
	float *d_rhs_uy; hipMalloc((void **)&d_rhs_uy, Nx*Ny*sizeof(float));
	
	//substrate information

	
	//forces
	float *d_ActForceX; hipMalloc((void **)&d_ActForceX, Nx*Ny*sizeof(float));
	float *d_ActForceY; hipMalloc((void **)&d_ActForceY, Nx*Ny*sizeof(float));
	//forces in forms of potential * grad(phi)
	float *d_ptl_force; hipMalloc((void **)&d_ptl_force, Nx*Ny*sizeof(float));
	
	//define gradient, laplacian, curvature, Gprime, absgrad
	float *d_dphix; hipMalloc((void **)&d_dphix, Nx*Ny*sizeof(float));
	float *d_dphiy; hipMalloc((void **)&d_dphiy, Nx*Ny*sizeof(float));
	float *d_phi_absgrad; hipMalloc((void **)&d_phi_absgrad, Nx*Ny*sizeof(float));
	float *d_phi_lap; hipMalloc((void **)&d_phi_lap, Nx*Ny*sizeof(float));

	float *d_phi_curv; hipMalloc((void **)&d_phi_curv, Nx*Ny*sizeof(float));
	float *d_ftd1_x; hipMalloc((void **)&d_ftd1_x, Nx*sizeof(float));
	float *d_ftd1_y; hipMalloc((void **)&d_ftd1_y, Ny*sizeof(float));
	float *d_ftd2_x; hipMalloc((void **)&d_ftd2_x, Nx*sizeof(float));
	float *d_ftd2_y; hipMalloc((void **)&d_ftd2_y, Ny*sizeof(float));

	float *d_xc_sin; hipMalloc((void **)&d_xc_sin, Nx*Ny*sizeof(float)); 
	float *d_xc_cos; hipMalloc((void **)&d_xc_cos, Nx*Ny*sizeof(float));
	//thrust::device_vector<float> d_sin(Nx*Ny);
	//thrust::device_vector<float> d_cos(Nx*Ny);




	//temporary buffers
	float *d_temp_buffer; hipMalloc((void **)&d_temp_buffer, Nx*Ny*sizeof(float)); //cublas buffer
	hipfftComplex *d_fftR2C_buffer; hipMalloc((void **)&d_fftR2C_buffer, (Nx / 2 + 1)*Ny*sizeof(hipfftComplex)); //cufft R2C buffer as fft result
	hipfftComplex *d_fftC2R_buffer; hipMalloc((void **)&d_fftC2R_buffer, (Nx / 2 + 1)*Ny*sizeof(hipfftComplex)); //as C2R buffer
	float *d_buffer_x; hipMalloc((void **)&d_buffer_x, Nx*Ny*sizeof(float));
	float *d_buffer_y; hipMalloc((void **)&d_buffer_y, Nx*Ny*sizeof(float));
	float *d_advect_buffer; hipMalloc((void **)&d_advect_buffer, Nx*Ny*sizeof(float));
	float *d_diffuse_buffer; hipMalloc((void **)&d_diffuse_buffer, Nx*Ny*sizeof(float));
	float *d_reaction_buffer; hipMalloc((void **)&d_reaction_buffer, Nx*Ny*sizeof(float));
	float *d_DiffRhoM; hipMalloc((void **)&d_DiffRhoM, Nx*Ny*sizeof(float)); //the rhoa-dependent rhom diffusion constant
	float *d_rhoAarea; hipMalloc((void **)&d_rhoAarea, sizeof(float)); //total active rhoA
	float *d_phiArea; hipMalloc((void **)&d_phiArea, sizeof(float)); //total area of phi
	float *d_rhoMoldArea; hipMalloc((void **)&d_rhoMoldArea, sizeof(float)); //total of rhoM old
	float *d_rhoMnewArea; hipMalloc((void **)&d_rhoMnewArea, sizeof(float)); //total of rhoM new
	float *d_bendcore; hipMalloc((void **)&d_bendcore,Nx*Ny*sizeof(float)); //the bending core

	float *d_unit_matrix; hipMalloc((void **)&d_unit_matrix,Nx*Ny*sizeof(float)); // matrix with all 1s
	
	//fft coefficients
	fftcoeffs(d_ftd1_x,d_ftd1_y, d_ftd2_x,d_ftd2_y);
	
	//handles
	hipfftPlan2d(&plan_R2C, Nx, Ny, HIPFFT_R2C);
	hipfftPlan2d(&plan_C2R, Nx, Ny, HIPFFT_C2R);
	hipblasCreate(&blashandle);
	hipblasSetPointerMode(blashandle, HIPBLAS_POINTER_MODE_DEVICE); //make the cublas return value to device
	hipblasCreate(&h_blashandle);

	//timer start
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	//initialize the global parameters
	//r0_cell = 6.0f; 
    
	Initialize(h_phi, h_RhoA, h_RhoM, h_ux, h_uy, h_x, h_y);
	//copy memory
	hipMemcpy(d_x, h_x, Nx*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, Ny*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_phi_old, h_phi, Nx*Ny*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_phi_new, h_phi, Nx*Ny*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_RhoAold, h_RhoA, Nx*Ny*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_RhoMold, h_RhoM, Nx*Ny*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_RhoAnew, h_RhoA, Nx*Ny*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_RhoMnew, h_RhoM, Nx*Ny*sizeof(float), hipMemcpyHostToDevice);	
	hipMemcpy(d_ux_old, h_ux, Nx*Ny*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_uy_old, h_uy, Nx*Ny*sizeof(float), hipMemcpyHostToDevice);
	//area size volume
	float A0;
	hipblasSasum(h_blashandle, Nx*Ny, d_phi_old, 1, &A0);
	A0 = A0 * dx * dy;
	//set unit matrix
	set_unit<<<blocks, threadsperblock>>>(d_unit_matrix, 1.0f, Nx, Ny);
	//float h_alpha = 1.0f; 
	//float h_beta = 1.0f;
	//error parameters
	float max_error;
	int iter_steps;

	int idx_maxerr_ux; 
	int idx_max_ux; 
	int idx_maxerr_uy; 
	int idx_max_uy; 
	float *d_max_error; hipMalloc((void **)&d_max_error,sizeof(float)); 

	//mass center
	grad_fft(d_dphix, d_dphiy, d_phi_old, d_ftd1_x, d_ftd1_y, d_fftR2C_buffer, d_fftC2R_buffer, Nx, Ny);
	absarray <<<blocks, threadsperblock >>>(d_phi_absgrad, d_dphix, d_dphiy, Nx, Ny);
	lap_fft(d_phi_lap, d_phi_old, d_ftd2_x, d_ftd2_y, d_fftR2C_buffer, d_fftC2R_buffer, Nx, Ny);

	//initialize fiber
	x_moc = 0.0f; y_moc = 0.0f;
	x_f_end = 0.0f; y_f_end = 0.0f;
	unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
	std::default_random_engine generator (seed);
	std::uniform_real_distribution<float> distribution(-Lx+0.5f, Lx-0.5f);

	float len_f = 0.0f;
	float  temp_p1, temp_p2, temp_p3; // record the period of 1 and 2 protrusion
	temp_p1 = 0.0f; temp_p2 = 0.0f; temp_p3 = 0.0f;
	while(len_f < 0.8f * Lx ){
		x_f_end = distribution(generator);
		y_f_end = distribution(generator);
		len_f = sqrt((x_f_end - x_moc)*(x_f_end - x_moc) + (y_f_end - y_moc)*(y_f_end - y_moc));
	}
	cylinder_phase<<<blocks, threadsperblock >>>(d_fiber_1, x_moc, y_moc, x_f_end, y_f_end, f_radius, d_x, d_y, Nx, Ny);
	Gphi<<<blocks, threadsperblock>>>(d_temp_buffer, d_phi_old, Nx, Ny);
	matrix_product<<<blocks, threadsperblock>>>(d_temp_buffer, d_fiber_1, d_temp_buffer, 1.0f, Nx, Ny);
	int maxIndex1;
	hipblasIsamax(h_blashandle, Nx * Ny, d_temp_buffer, 1, &maxIndex1);
	disc_phase<<<blocks, threadsperblock>>>(d_disc_1, maxIndex1, h_disc_r, d_x, d_y, Nx, Ny);
	ran_per_1 = h_period + (distribution(generator) / Lx) * var_p;

	//len_f = 0.0f;
	//while(len_f < 0.8f * Lx ){
	//	x_f_end = distribution(generator);
	//	y_f_end = distribution(generator);
	//	len_f = sqrt((x_f_end - x_moc)*(x_f_end - x_moc) + (y_f_end - y_moc)*(y_f_end - y_moc));
	//}
	//have the opposite point
	x_f_end = 2.0f * x_moc - x_f_end;
	y_f_end = 2.0f * y_moc - y_f_end;
	cylinder_phase<<<blocks, threadsperblock >>>(d_fiber_2, x_moc, y_moc, x_f_end, y_f_end, f_radius, d_x, d_y, Nx, Ny);


	Gphi<<<blocks, threadsperblock>>>(d_temp_buffer, d_phi_old, Nx, Ny);
	matrix_product<<<blocks, threadsperblock>>>(d_temp_buffer, d_fiber_2, d_temp_buffer, 1.0f, Nx, Ny);	
	int maxIndex2;
	hipblasIsamax(h_blashandle, Nx * Ny, d_temp_buffer, 1, &maxIndex2);
	disc_phase<<<blocks, threadsperblock>>>(d_disc_2, maxIndex2, h_disc_r, d_x, d_y, Nx, Ny);
	ran_per_2 = h_period + (distribution(generator) / Lx) * var_p;

	len_f = 0.0f;
	while(len_f < 0.8f * Lx ){
		x_f_end = distribution(generator);
		y_f_end = distribution(generator);
		len_f = sqrt((x_f_end - x_moc)*(x_f_end - x_moc) + (y_f_end - y_moc)*(y_f_end - y_moc));
	}
	cylinder_phase<<<blocks, threadsperblock >>>(d_fiber_3, x_moc, y_moc, x_f_end, y_f_end, f_radius, d_x, d_y, Nx, Ny);
	Gphi<<<blocks, threadsperblock>>>(d_temp_buffer, d_phi_old, Nx, Ny);
	matrix_product<<<blocks, threadsperblock>>>(d_temp_buffer, d_fiber_3, d_temp_buffer, 1.0f, Nx, Ny);
	int maxIndex3;
	hipblasIsamax(h_blashandle, Nx * Ny, d_temp_buffer, 1, &maxIndex3);
	disc_phase<<<blocks, threadsperblock>>>(d_disc_3, maxIndex3, h_disc_r, d_x, d_y, Nx, Ny);
	ran_per_3 = h_period / 2.0f + (distribution(generator) / Lx) * var_p;


	for (int steps = 0; steps<(int)(max_time / dt); steps++){

		//curv <<<blocks, threadsperblock >>>(d_phi_curv, d_dphix, d_dphiy, d_phi_absgrad, Nx, Ny, dx,dy);
		curv <<<blocks, threadsperblock >>>(d_phi_curv, d_phi_old, d_phi_absgrad, Nx, Ny, dx,dy);
		//solve phi
		phievolve <<<blocks, threadsperblock >>>(d_phi_new, d_phi_old, d_dphix, d_dphiy, d_phi_absgrad, d_phi_lap, d_phi_curv, d_ux_old, d_uy_old, Nx, Ny, dt);

		//update derivatives
		grad_fft(d_dphix, d_dphiy, d_phi_new, d_ftd1_x, d_ftd1_y, d_fftR2C_buffer, d_fftC2R_buffer, Nx, Ny);
		absarray <<<blocks, threadsperblock >>>(d_phi_absgrad, d_dphix, d_dphiy, Nx, Ny);
		lap_fft(d_phi_lap, d_phi_new, d_ftd2_x, d_ftd2_y, d_fftR2C_buffer, d_fftC2R_buffer, Nx, Ny);

		
		//update rhoA and rhoM
		//rhoA advection
		//div_advect_pre<<<blocks, threadsperblock >>>(d_div_x, d_div_y, d_phi_old, d_RhoAold, d_ux_old, d_uy_old, Nx, Ny);
		//div_fft(d_advect_buffer, d_div_x, d_div_y, d_ftd1_x, d_ftd1_y, d_fftR2C_buffer, d_fftC2R_buffer, Nx, Ny);
		//rhoA diffusion        	
		//div_pst_fd<<<blocks, threadsperblock >>>(d_diffuse_buffer, d_phi_old, d_RhoAold, h_diffRhoA, dx, dy, Nx, Ny);
		//rhoA reaction
		hipblasSasum(blashandle, Nx * Ny, d_phi_old, 1, d_phiArea);
		//matrix_product<<<blocks, threadsperblock >>>(d_temp_buffer, d_RhoAold, d_phi_old, 1.0f, Nx, Ny);
		//hipblasSasum(blashandle, Nx * Ny, d_temp_buffer, 1, d_rhoAarea);
		//reactionRhoA<<<blocks, threadsperblock >>>(d_reaction_buffer, d_phi_old, d_RhoAold, d_rhoAarea, d_phiArea, h_RhoAtot, h_kbRhoA, h_K2aRhoA, h_kaRhoA, h_kcRhoA, dx, dy, Nx, Ny);
		//time evolve
		//RD_evolve<<<blocks, threadsperblock >>>(d_RhoAnew, d_phi_new, d_phi_old, d_RhoAold, d_advect_buffer, d_diffuse_buffer, d_reaction_buffer, dt, lamda, Nx, Ny);
		//rhoa_right<<<blocks, threadsperblock>>>(d_RhoAold, d_phi_old, h_xsin, h_xcos, d_x, Lx, Nx, Ny);
		//rhoa_right<<<blocks, threadsperblock>>>(d_RhoAnew, d_phi_new, h_xsin, h_xcos, d_x, Lx, Nx, Ny);
		temp_p1 = temp_p1 + dt;
		temp_p2 = temp_p2 + dt;
		temp_p3 = temp_p3 + dt;

		if(temp_p1 > ran_per_1){
			len_f = 0.0f;
			while(len_f < 0.8f * Lx ){
				x_f_end = distribution(generator);
				y_f_end = distribution(generator);
				len_f = sqrt((x_f_end - x_moc)*(x_f_end - x_moc) + (y_f_end - y_moc)*(y_f_end - y_moc));
			}
			//get the dis shape rhoa region centered at max(fiber*boudary)
			cylinder_phase<<<blocks, threadsperblock >>>(d_fiber_1, x_moc, y_moc, x_f_end, y_f_end, f_radius, d_x, d_y, Nx, Ny);
			ran_per_1 = h_period + (distribution(generator) / Lx) * var_p;
			
			// this is the opposite side of limE
			//x_f_end = 2.0f * x_moc - x_f_end;
			//y_f_end = 2.0f * y_moc - y_f_end;
			//cylinder_phase<<<blocks, threadsperblock >>>(d_fiber_2, x_moc, y_moc, x_f_end, y_f_end, f_radius, d_x, d_y, Nx, Ny);
			//ran_per_2 = h_period + (distribution(generator) / Lx) * var_p;
		
			temp_p1 = 0.0f;
		}

		if(temp_p2 >= ran_per_2){
			len_f = 0.0f;
			while(len_f < 0.8f * Lx ){
				x_f_end = distribution(generator);
				y_f_end = distribution(generator);
				len_f = sqrt((x_f_end - x_moc)*(x_f_end - x_moc) + (y_f_end - y_moc)*(y_f_end - y_moc));
			}
			cylinder_phase<<<blocks, threadsperblock >>>(d_fiber_2, x_moc, y_moc, x_f_end, y_f_end, f_radius, d_x, d_y, Nx, Ny);
			ran_per_2 = h_period + (distribution(generator) / Lx) * var_p;
			temp_p2 = 0.0f;
		}

		//myosin patch changes when time exceeds random value
		if( temp_p3 >= ran_per_3){
			len_f = 0.0f;
			while(len_f < 0.8f * Lx ){
				x_f_end = distribution(generator);
				y_f_end = distribution(generator);
				len_f = sqrt((x_f_end - x_moc)*(x_f_end - x_moc) + (y_f_end - y_moc)*(y_f_end - y_moc));
			}
			cylinder_phase<<<blocks, threadsperblock >>>(d_fiber_3, x_moc, y_moc, x_f_end, y_f_end, f_radius, d_x, d_y, Nx, Ny);
			ran_per_3 = h_period / 2.0f + (distribution(generator) / Lx) * var_p;
			temp_p3 = 0.0f;
		}

		if (steps % 100 == 0){
			//stepping the rhoa rhom disc every 100 steps (0.2s)
			Gphi<<<blocks, threadsperblock>>>(d_temp_buffer, d_phi_old, Nx, Ny);
			matrix_product<<<blocks, threadsperblock>>>(d_temp_buffer, d_fiber_1, d_temp_buffer, 1.0f, Nx, Ny);
			hipblasIsamax(h_blashandle, Nx * Ny, d_temp_buffer, 1, &maxIndex1);
			disc_phase<<<blocks, threadsperblock>>>(d_disc_1, maxIndex1, h_disc_r, d_x, d_y, Nx, Ny);

			Gphi<<<blocks, threadsperblock>>>(d_temp_buffer, d_phi_old, Nx, Ny);
			matrix_product<<<blocks, threadsperblock>>>(d_temp_buffer, d_fiber_2, d_temp_buffer, 1.0f, Nx, Ny);
			hipblasIsamax(h_blashandle, Nx * Ny, d_temp_buffer, 1, &maxIndex2);
			disc_phase<<<blocks, threadsperblock>>>(d_disc_2, maxIndex2, h_disc_r, d_x, d_y, Nx, Ny);

			Gphi<<<blocks, threadsperblock>>>(d_temp_buffer, d_phi_old, Nx, Ny);
			matrix_product<<<blocks, threadsperblock>>>(d_temp_buffer, d_fiber_3, d_temp_buffer, 1.0f, Nx, Ny);
			hipblasIsamax(h_blashandle, Nx * Ny, d_temp_buffer, 1, &maxIndex3);
			disc_phase<<<blocks, threadsperblock>>>(d_disc_3, maxIndex3, h_disc_r, d_x, d_y, Nx, Ny);
		}

		rhoa_am<<<blocks, threadsperblock>>>(d_RhoAold, d_RhoMold, d_phi_old, d_disc_1, d_disc_2, d_disc_3, steps * dt, ran_per_1, ran_per_2, Nx, Ny);
		rhoa_am<<<blocks, threadsperblock>>>(d_RhoAnew, d_RhoMnew, d_phi_new, d_disc_1, d_disc_2, d_disc_3, steps * dt, ran_per_1, ran_per_2, Nx, Ny);

		//rhoa_osci<<<blocks, threadsperblock>>>(d_RhoAold, d_RhoMold, d_phi_old, h_xsin, h_xcos, steps * dt, A_peri, d_x, Lx, Nx, Ny);
		//rhoa_osci<<<blocks, threadsperblock>>>(d_RhoAnew, d_RhoMnew, d_phi_new, h_xsin, h_xcos, steps * dt, A_peri, d_x, Lx, Nx, Ny);
		
		//rhoM advection
		//div_advect_pre<<<blocks, threadsperblock >>>(d_div_x, d_div_y, d_phi_old, d_RhoMold, d_ux_old, d_uy_old, Nx, Ny);
		//div_fft(d_advect_buffer, d_div_x, d_div_y, d_ftd1_x, d_ftd1_y, d_fftR2C_buffer, d_fftC2R_buffer, Nx, Ny);
		//rhoM diffusion
		//diffRhoM<<<blocks, threadsperblock >>>(d_DiffRhoM, d_phi_old, d_RhoAold, h_diffRhoM0, h_KdRhoM, Nx, Ny);
		//div_pst_fd<<<blocks, threadsperblock >>>(d_diffuse_buffer, d_DiffRhoM, d_RhoMold, 1.0f, dx, dy, Nx, Ny);
		//no rhoM reaction
		//RD_evolve<<<blocks, threadsperblock >>>(d_RhoMnew, d_phi_new, d_phi_old, d_RhoMold, d_advect_buffer, d_diffuse_buffer, dt, lamda, Nx, Ny);
		//normRhoM rhoM
		//matrix_product<<<blocks, threadsperblock>>>(d_temp_buffer, d_RhoMold, d_phi_old, dx * dy, Nx, Ny);
		//hipblasSasum(blashandle, Nx * Ny, d_temp_buffer, 1, d_rhoMoldArea);
		//matrix_product<<<blocks, threadsperblock>>>(d_temp_buffer, d_RhoMnew, d_phi_new, dx * dy, Nx, Ny);
		//hipblasSasum(blashandle, Nx * Ny, d_temp_buffer, 1, d_rhoMnewArea);
		//normRhoM<<<blocks, threadsperblock>>>(d_RhoMnew, d_rhoMnewArea, d_rhoMoldArea, Nx, Ny);

		//only rhoA at the right
		

		//update forces
		bend_core<<<blocks, threadsperblock>>>(d_bendcore, d_phi_lap, d_phi_new, Nx, Ny);
		lap_fft(d_bendcore, d_bendcore, d_ftd2_x, d_ftd2_y, d_fftR2C_buffer, d_fftC2R_buffer, Nx, Ny);
		potential_force<<<blocks, threadsperblock >>>(d_ptl_force, d_phi_new, d_phi_lap, d_bendcore, d_phiArea, A0, dx, dy, Nx, Ny);
		
		//active force
		activeForce<<<blocks, threadsperblock>>>(d_div_x, d_div_y, 1, d_phi_new, h_etaRhoA, h_etaRhoM, d_RhoAnew, d_RhoMnew, d_dphix, d_dphiy, Nx, Ny);
		div_fft(d_ActForceX, d_div_x, d_div_y, d_ftd1_x, d_ftd1_y, d_fftR2C_buffer, d_fftC2R_buffer, Nx, Ny);

		activeForce<<<blocks, threadsperblock>>>(d_div_x, d_div_y, 2, d_phi_new, h_etaRhoA, h_etaRhoM, d_RhoAnew, d_RhoMnew, d_dphix, d_dphiy, Nx, Ny);
		div_fft(d_ActForceY, d_div_x, d_div_y, d_ftd1_x, d_ftd1_y, d_fftR2C_buffer, d_fftC2R_buffer, Nx, Ny);
		
		max_error = 10.0f;
		iter_steps = 0;
		while(max_error > poisson_err_limit && iter_steps < poisson_max_steps){
		    //grad(ux)
		    grad_fft(d_duxdx, d_duxdy, d_ux_old, d_ftd1_x, d_ftd1_y, d_fftR2C_buffer, d_fftC2R_buffer, Nx, Ny);
		    //grad(uy)
		    grad_fft(d_duydx, d_duydy, d_uy_old, d_ftd1_x, d_ftd1_y, d_fftR2C_buffer, d_fftC2R_buffer, Nx, Ny);
		    //grad(uz)
			//vel_div_x
		    div_vel_pre<<<blocks, threadsperblock >>>(d_div_x, d_div_y, 1, d_phi_new,  nu_vis, d_duxdx, d_duxdy, d_duydx, d_duydy, Nx, Ny);
		    div_fft(d_buffer_x, d_div_x, d_div_y, d_ftd1_x, d_ftd1_y, d_fftR2C_buffer, d_fftC2R_buffer, Nx, Ny);
		    //vel_div_y
		    div_vel_pre<<<blocks, threadsperblock >>>(d_div_x, d_div_y, 2, d_phi_new,  nu_vis, d_duxdx, d_duxdy, d_duydx, d_duydy, Nx, Ny);
		    div_fft(d_buffer_y, d_div_x, d_div_y, d_ftd1_x, d_ftd1_y, d_fftR2C_buffer, d_fftC2R_buffer, Nx, Ny);
		   
		    rhs_poisson<<<blocks, threadsperblock >>>(d_rhs_ux, d_rhs_uy, d_ptl_force, d_phi_new, d_dphix, d_dphiy, d_buffer_x, d_buffer_y, d_ActForceX, d_ActForceY, d_RhoAnew, d_RhoMnew, d_ux_old, d_uy_old, Nx, Ny);

		    poisson_sol(d_ux_new, d_rhs_ux, xi_fric, nu_vis*exceed_val, d_ftd2_x, d_ftd2_y, d_fftR2C_buffer, d_fftC2R_buffer, Nx, Ny);
		    poisson_sol(d_uy_new, d_rhs_uy, xi_fric, nu_vis*exceed_val, d_ftd2_x, d_ftd2_y, d_fftR2C_buffer, d_fftC2R_buffer, Nx, Ny);

		    //modify ux uy uz in regions of phi < 0.0001 not done
		    
		    //get error
		    minus_matrix<<<blocks, threadsperblock >>>(d_buffer_x, d_ux_new, d_ux_old, Nx, Ny);
			minus_matrix<<<blocks, threadsperblock >>>(d_buffer_y, d_uy_new, d_uy_old, Nx, Ny);
			hipblasIsamax(h_blashandle, Nx*Ny, d_buffer_x, 1, &idx_maxerr_ux);
			hipblasIsamax(h_blashandle, Nx*Ny, d_ux_new, 1, &idx_max_ux);
			hipblasIsamax(h_blashandle, Nx*Ny, d_buffer_y, 1, &idx_maxerr_uy);
			hipblasIsamax(h_blashandle, Nx*Ny, d_uy_new, 1, &idx_max_uy);
			get_error<<<1, 1>>>(d_max_error, d_ux_new, d_ux_old, d_uy_new, d_uy_old, idx_maxerr_ux, idx_max_ux, idx_maxerr_uy, idx_max_uy, Nx, Ny);
			
			hipMemcpy(&max_error, d_max_error, sizeof(float), hipMemcpyDeviceToHost);

		    iter_steps++;

		    hipblasScopy(blashandle, Nx*Ny, d_ux_new, 1, d_ux_old, 1);
		    hipblasScopy(blashandle, Nx*Ny, d_uy_new, 1, d_uy_old, 1);

		}
		if(max_error > poisson_err_limit){
		    printf("step %f with %d iterations wrong err %f\n", steps * dt, iter_steps, max_error);
		    break;
		}

		//record center
		if(steps % 500 == 0){
			xdir_center<<<blocks, threadsperblock>>>(d_xc_sin, d_xc_cos, d_phi_new, d_phiArea, d_y, Ly, Nx, Ny);
			thrust::device_ptr<float> d_sin = thrust::device_pointer_cast(d_xc_sin);
			h_xsin = thrust::reduce(d_sin, d_sin + Nx*Ny);
			thrust::device_ptr<float> d_cos = thrust::device_pointer_cast(d_xc_cos);
			h_xcos = thrust::reduce(d_cos, d_cos + Nx*Ny);
			x_moc = atan2(h_xsin, h_xcos)/M_PI*Lx;

			ydir_center<<<blocks, threadsperblock>>>(d_xc_sin, d_xc_cos, d_phi_new, d_phiArea, d_y, Ly, Nx, Ny);
			d_sin = thrust::device_pointer_cast(d_xc_sin);
			h_xsin = thrust::reduce(d_sin, d_sin + Nx*Ny);
			d_cos = thrust::device_pointer_cast(d_xc_cos);
			h_xcos = thrust::reduce(d_cos, d_cos + Nx*Ny);
			y_moc = atan2(h_xsin, h_xcos)/M_PI*Ly;

			//h_xsin = 0.0f; h_xcos = 0.0f;
			//xdir_center<<<blocks, threadsperblock>>>(d_xc_sin, d_xc_cos, d_phi_new, d_phiArea, d_x, Lx, Nx, Ny);
			//hipblasSgemm(h_blashandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, 1, Nx*Ny, &h_alpha, d_xc_sin, 1, d_unit_matrix, Nx*Ny, &h_beta, &h_xsin, 1);
			//hipblasSgemm(h_blashandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, 1, Nx*Ny, &h_alpha, d_xc_cos, 1, d_unit_matrix, Nx*Ny, &h_beta, &h_xcos, 1);
			//x_moc = atan2(h_xsin, h_xcos)/M_PI*Lx;

			//h_xsin = 0.0f; h_xcos = 0.0f;
			//ydir_center<<<blocks, threadsperblock>>>(d_xc_sin, d_xc_cos, d_phi_new, d_phiArea, d_y, Ly, Nx, Ny);
			//hipblasSgemm(h_blashandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, 1, Nx*Ny, &h_alpha, d_xc_sin, 1, d_unit_matrix, Nx*Ny, &h_beta, &h_xsin, 1);
			//hipblasSgemm(h_blashandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, 1, Nx*Ny, &h_alpha, d_xc_cos, 1, d_unit_matrix, Nx*Ny, &h_beta, &h_xcos, 1);
			//y_moc = atan2(h_xsin, h_xcos)/M_PI*Ly;

			fprintf(ft, "%f %f %f\n", steps*dt, x_moc, y_moc);
		}

		if(steps % (int)(record_itvl/dt)==1){
			char phi_name[50];sprintf(phi_name,"phi_profile_%d.txt", file_record);
			hipMemcpy(h_phi, d_phi_old, Nx*Ny*sizeof(float), hipMemcpyDeviceToHost);
			writetofile(phi_name, h_phi, Nx, Ny);

			char rhoA_name[50];sprintf(rhoA_name,"rhoA_profile_%d.txt",file_record);
			hipMemcpy(h_RhoA, d_RhoAold, Nx*Ny*sizeof(float), hipMemcpyDeviceToHost);
			writetofile(rhoA_name, h_RhoA, Nx, Ny);

			char rhoM_name[50];sprintf(rhoM_name,"rhoM_profile_%d.txt",file_record);
			hipMemcpy(h_RhoM, d_RhoMold, Nx*Ny*sizeof(float), hipMemcpyDeviceToHost);
			writetofile(rhoM_name, h_RhoM, Nx, Ny);

			char ux_name[50];sprintf(ux_name,"ux_profile_%d.txt",file_record);
			hipMemcpy(h_ux, d_ux_old, Nx*Ny*sizeof(float), hipMemcpyDeviceToHost);
			writetofile(ux_name, h_ux, Nx, Ny);

			char uy_name[50];sprintf(uy_name,"uy_profile_%d.txt",file_record);
			hipMemcpy(h_uy, d_uy_old, Nx*Ny*sizeof(float), hipMemcpyDeviceToHost);
			writetofile(uy_name, h_uy, Nx, Ny);
			
			file_record++;

		}

		//swap old and new
		hipblasScopy(blashandle, Nx*Ny, d_phi_new, 1, d_phi_old, 1);
		//hipblasScopy(blashandle, Nx*Ny, d_RhoAnew, 1, d_RhoAold, 1);
		//hipblasScopy(blashandle, Nx*Ny, d_RhoMnew, 1, d_RhoMold, 1);

	}
	
	//final record of the shape
	char phi_name[50]; sprintf(phi_name,"phi_profile.txt", file_record);
	hipMemcpy(h_phi, d_phi_old, Nx*Ny*sizeof(float), hipMemcpyDeviceToHost);
	writetofile(phi_name, h_phi, Nx, Ny);

	char rhoA_name[50]; sprintf(rhoA_name,"rhoA_profile.txt",file_record);
	hipMemcpy(h_RhoA, d_RhoAold, Nx*Ny*sizeof(float), hipMemcpyDeviceToHost);
	writetofile(rhoA_name, h_RhoA, Nx, Ny);

	char rhoM_name[50]; sprintf(rhoM_name,"rhoM_profile.txt",file_record);
	hipMemcpy(h_RhoM, d_RhoMold, Nx*Ny*sizeof(float), hipMemcpyDeviceToHost);
	writetofile(rhoM_name, h_RhoM, Nx, Ny);

	char ux_name[50];sprintf(ux_name,"ux_profile.txt",file_record);
	hipMemcpy(h_ux, d_ux_old, Nx*Ny*sizeof(float), hipMemcpyDeviceToHost);
	writetofile(ux_name, h_ux, Nx, Ny);

	char uy_name[50];sprintf(uy_name,"uy_profile.txt",file_record);
	hipMemcpy(h_uy, d_uy_old, Nx*Ny*sizeof(float), hipMemcpyDeviceToHost);
	writetofile(uy_name, h_uy, Nx, Ny);
	
	//destroy plan and handles
	hipfftDestroy(plan_R2C);
	hipfftDestroy(plan_C2R);
	hipblasDestroy(blashandle);
	hipblasDestroy(h_blashandle);
	fclose(ft);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsetime;
	hipEventElapsedTime(&elapsetime, start, stop);

	std::printf("time need: %f s \n", elapsetime / 1000);

	return 0;
}

void Initialize(float* phi, float *rhoA, float *rhoM, float* ux, float* uy, float* x, float* y){
	for (int i = 0; i < Nx; i++)x[i] = -Lx + 2 * Lx / (float)Nx*(float)i;
	for (int i = 0; i < Ny; i++)y[i] = -Ly + 2 * Ly / (float)Ny*(float)i;
	
	for (int i = 0; i < Nx; i++){
		for (int j = 0; j < Ny; j++){
			float dis = sqrt(x[i]*x[i] + y[j]*y[j]);
			int index = i + j * Nx;
			phi[index]=0.5f + 0.5f * tanh(3.0f*(r0_cell-dis) / h_epsilon);

			rhoA[index] = 0.0f;
			if(x[i] > 0.0f){
				rhoA[index] = rhoAinitial * phi[index];
			}
			rhoM[index] = rhoMinitial * phi[index];
			ux[index] = 0.0f;
			uy[index] = 0.0f;			
		}
	}
}

void writetofile(const char *name, float *u, int Nx, int Ny){
	FILE *fp = fopen(name, "w+");
		for (int j = 0; j < Ny; j++){
			for (int i = 0; i < Nx; i++){
				fprintf(fp, "%f ", u[i + j*Nx]);
			}
			fprintf(fp, "\n");
		}
	
	fclose(fp);
}

__global__ void phievolve(float *phi_new, float *phi_old, float *grad_x, float *grad_y, float *absdphi, float *lap_phi, float *curv, float *ux, float *uy, int Nx, int Ny, float dt){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx;

	float dG = 36.0f * phi_old[index] * (1.0f - phi_old[index]) * (1.0f - 2.0f * phi_old[index]);

	phi_new[index] = phi_old[index] + dt*(-ux[index] * grad_x[index] - uy[index] * grad_y[index] + d_Gamma*(d_epsilon*lap_phi[index] - dG / d_epsilon + curv[index] * d_epsilon * absdphi[index]));
}

__global__ void div_vel_pre(float *div_x, float *div_y, int dim, float *phi, float nu, float *duxdx, float *duxdy, float *duydx, float *duydy, int Nx, int Ny){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx;

    	if(dim == 1){
        	div_x[index] = 2.0f * nu * phi[index] * duxdx[index] - nu * d_exceed_val * duxdx[index]; //sigma_xx
        	div_y[index] = nu * phi[index] * (duxdy[index] + duydx[index]) - nu * d_exceed_val * duxdy[index]; //sigma_xy
    	}
    	if(dim == 2){
        	div_x[index] = nu * phi[index] *(duydx[index] + duxdy[index]) - nu * d_exceed_val * duydx[index]; //sigma_yx
        	div_y[index] = 2.0f * nu * phi[index] * duydy[index] - nu * d_exceed_val * duydy[index]; //sigma_yy
    	}
}


__global__ void potential_force(float *output, float *phi, float *lap_phi, float *bendcore, float *phiarea, float A0, float dx, float dy, int Nx, int Ny){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx;
	float dGphi = 36.0f * phi[index] * (1.0f - phi[index]) * (1.0f - 2.0f*phi[index]);
	float ddGphi = 36.0f*(1.0f - 6.0f*phi[index] + 6.0f*phi[index] * phi[index]);
	float core = lap_phi[index] - dGphi / d_epsilon / d_epsilon;

	float current_area = *phiarea * dx * dy; 
	float delta_area = 0.0f;
	if(current_area < d_areaMin)	
		delta_area = current_area - d_areaMin;
	if(current_area > d_areaMax)
		delta_area = current_area - d_areaMax;

	output[index] = -d_tension * d_epsilon * core + d_bending * d_epsilon * (bendcore[index] - ddGphi * core / d_epsilon / d_epsilon) + d_Mv * delta_area;
}

__global__ void rhs_poisson(float *rhs_x, float *rhs_y, float *ptl_force, float *phi, float *dphix, float *dphiy, float *vis_x, float *vis_y, float *act_x, float *act_y, float *rhoA, float *rhoM, float *ux, float *uy, int Nx, int Ny){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx;
	//float Gphi = 18.0f * phi[index] * phi[index] * (1.0f - phi[index]) * (1.0f - phi[index]) / d_epsilon;
	//float delta_phi = ux[index] * dphix[index] + uy[index] * dphiy[index];
	//if(delta_phi < 0.0f) delta_phi = 0.0f;
	rhs_x[index] = ptl_force[index] * dphix[index] + act_x[index] + vis_x[index] - d_xiM * rhoM[index] * ux[index] * phi[index];
	rhs_y[index] = ptl_force[index] * dphiy[index] + act_y[index] + vis_y[index] - d_xiM * rhoM[index] * uy[index] * phi[index];
}



__global__ void get_error(float *max_error, float *ux_new, float *ux_old, float *uy_new, float *uy_old, int idx_eux, int idx_ux, int idx_euy, int idx_uy, int Nx, int Ny){
	float err_ux = abs(ux_new[idx_eux - 1] - ux_old[idx_eux - 1]) / abs(ux_new[idx_ux - 1]);
	float err_uy = abs(uy_new[idx_euy - 1] - uy_old[idx_euy - 1]) / abs(uy_new[idx_uy - 1]);

	*max_error = MAX(err_ux, err_uy);
}


__global__ void div_advect_pre(float *div_x, float *div_y, float *dnsty, float *rho, float *ux, float *uy, int Nx, int Ny){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx;
	//advection on the interface
	div_x[index] = dnsty[index] * rho[index] * ux[index];
	div_y[index] = dnsty[index] * rho[index] * uy[index];
}


__global__ void RD_evolve(float *rho_new, float *dnsty_new, float *dnsty_old, float *rho_old, float *advect, float *diffuse, float *react, float dt, float lamda, int Nx, int Ny){
	//with reaction terms
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx;

	rho_new[index] = 0.0f;
	if(dnsty_old[index] > lamda){
		rho_new[index] = (2.0f * dnsty_old[index] - dnsty_new[index]) / dnsty_old[index] * rho_old[index] + dt / dnsty_old[index] *(diffuse[index] - advect[index] + react[index]);           
	}
}

__global__ void RD_evolve(float *rho_new, float *dnsty_new, float *dnsty_old, float *rho_old, float *advect, float *diffuse, float dt, float lamda, int Nx, int Ny){
	//without reaction terms
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx;

	rho_new[index] = 0.0f;
	if(dnsty_old[index] > lamda){
		rho_new[index] = (2.0f * dnsty_old[index] - dnsty_new[index]) / dnsty_old[index] * rho_old[index] + dt / dnsty_old[index] *(diffuse[index] - advect[index]);           
	}

}

__global__ void normRhoM(float *rho, float *rho_area_new, float *rho_are_old, int Nx, int Ny){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx;

	rho[index] = rho[index] * (*rho_are_old) / (*rho_area_new);
}


__global__ void diffRhoM(float *output, float *phi, float *rhoa, float Dm0, float Kd, int Nx, int Ny){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx;

	output[index] = phi[index] * Dm0 / (1.0f + rhoa[index] / Kd);
}

__global__ void reactionRhoA(float *output, float *phi, float *rhoA, float *rhoAarea, float *phiarea, float rhoAtot, float kb, float K2a, float ka,    float kc, float dx, float dy,  int Nx, int Ny){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx;
	
	float rhoAcyt = (rhoAtot - (*rhoAarea) * dx * dy) / ((*phiarea) * dx * dy);
	output[index] = phi[index] * (kb * (rhoA[index] * rhoA[index] / (K2a * K2a + rhoA[index] * rhoA[index]) + ka) * rhoAcyt - kc * rhoA[index]);

}

__global__ void bend_core(float *core, float *lap_phi, float *phi, int Nx, int Ny){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx;

	if (i < Nx && j < Ny){
		core[index]=lap_phi[index]-36.0f*phi[index]*(1.0f-phi[index])*(1.0f-2.0f*phi[index])/d_epsilon/d_epsilon;
	}
}

__global__ void activeForce(float *div_x, float *div_y, int dir, float *phi, float etaRhoA, float etaRhoM, float *rhoA, float *rhoM, float *dphix, float *dphiy, int Nx, int Ny){
	//active force by rhoA and rhoM
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx;
	//x-dir
	if(dir == 1){
		div_x[index] = -etaRhoA * d_epsilon * phi[index] * rhoA[index] * dphix[index] * dphix[index] + etaRhoM * phi[index] * rhoM[index];
		div_y[index] = -etaRhoA * d_epsilon * phi[index] * rhoA[index] * dphix[index] * dphiy[index];
	}
	//y-dir
	if(dir == 2){
		div_x[index] = -etaRhoA * d_epsilon * phi[index] * rhoA[index] * dphix[index] * dphiy[index];
		div_y[index] = -etaRhoA * d_epsilon * phi[index] * rhoA[index] * dphiy[index] * dphiy[index] + etaRhoM * phi[index] * rhoM[index];
	}
}


/*--------------------------------------------------------------------------------------*/
/*Utility functions*/
/*--------------------------------------------------------------------------------------*/
__global__ void matrix_product(float *output, float *input1, float *input2, float alpha, int Nx, int Ny){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx;

	output[index] = alpha * input1[index] * input2[index];
}

__global__ void add3matrix(float *output, float *input1, float *input2, float *input3){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx;
	output[index] = input1[index] + input2[index] + input3[index];
}

__global__ void add2matrix(float *output, float *input1, float *input2){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx;
	output[index] = input1[index] + input2[index];
}

__global__ void absarray(float *absay, float *ax, float *ay, int Nx, int Ny){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j*Nx;
	
    absay[index] = sqrt(ax[index] * ax[index] + ay[index] * ay[index]);
}

__global__ void minus_matrix(float *output, float *minuend, float *substractor, int Nx, int Ny){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx;

	output[index] = minuend[index] - substractor[index];
}

__global__ void xdir_center(float *xc_sin, float *xc_cos, float *phi, float *phi_area, float *x, float Lx, int m, int n){
	//return the matrix of phi*sin(x*pi/Lx), phi*cos(x*phi/Lx)
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j*blockDim.x*gridDim.x;
	if(i< m && j<n){
		xc_sin[index] = phi[index] / (*phi_area) * sin(x[i] * M_PI / Lx);
		xc_cos[index] = phi[index] / (*phi_area) * cos(x[i] * M_PI / Lx);
	}
}

__global__ void ydir_center(float *xc_sin, float *xc_cos, float *phi, float *phi_area, float *y, float Lx, int m, int n){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j*blockDim.x*gridDim.x;
	if(i< m && j<n){
		xc_sin[index] = phi[index] / (*phi_area) * sin(y[j] * M_PI / Lx);
		xc_cos[index] = phi[index] / (*phi_area) * cos(y[j] * M_PI / Lx);
	}
}

__global__ void xdir_center(float *xc_sin, float *xc_cos, float *phi, float *phi_area, float *x, float Lx, int label, int m, int n){
	//return the matrix of phi*sin(x*pi/Lx), phi*cos(x*phi/Lx)
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j*blockDim.x*gridDim.x;
	if(i< m && j<n){
		if(label == 1){
			xc_sin[index] = phi[index] / (*phi_area) * sin(x[i] * M_PI / Lx);
			xc_cos[index] = phi[index] / (*phi_area) * cos(x[i] * M_PI / Lx);
		}else{
			xc_sin[index] = phi[index] / (*phi_area) * sin(x[j] * M_PI / Lx);
			xc_cos[index] = phi[index] / (*phi_area) * cos(x[j] * M_PI / Lx);
		}
	}
}

__global__ void rhoa_right(float *rhoa, float *phi, float x_sin, float x_cos, float *x, float Lx, int m, int n){
	//put rhoa at the right half plane of mass centern
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j*m;
	if( i < m && j < n){
		rhoa[index] = 0.0f; 
		if(sin(x[i]*M_PI/Lx) * x_cos - cos(x[i]*M_PI/Lx)*x_sin > -0.5f && phi[index] > 1e-4){
			rhoa[index] = 1.0f;
		}
	}
}

__global__ void rhoa_am(float *rhoa, float *rhom, float *phi, float *fiber1, float *fiber2, float *fiber3, float t, float period1, float period2, int m, int n){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j*m;
	if( i < m && j < n){
		float signal1 = cos(2.0f * M_PI * t / period1);
		float signal2 = cos(2.0f * M_PI * t / period2);
		//float signal2a = sin(M_PI * t / period2);
		//float signal2m = sin(M_PI * t / period2);
		//rhoa[index] = fiber1[index] * (1.0f + signal1) / 2.0f;
		float sign1 = (float)(sin(2.0f * M_PI * t / period1) <= 0.0f); // 1 for positive and 0 for negative
		float sign2 = (float)(sin(2.0f * M_PI * t / period2) <= 0.0f);
		
		float Gphi = 18.0 * phi[index] * phi[index] * (1.0f - phi[index]) * (1.0f - phi[index]) / d_epsilon;
		//if(phi[index]<0.98f) Gphi = 1.0f;
		
		rhoa[index] = fiber1[index] *  (1.0f - signal1) / 2.0f + fiber2[index] * (1.0f - signal2) / 2.0f;
		//rhoa[index] = fiber1[index] * sign1 + fiber2[index] * (1.0f - sign2);
		//rhom[index] = fiber1[index] * (1.0f - signal1) / 2.0f * (1.0f - sign1) + fiber2[index] * (1.0f + signal2) / 2.0f * sign2; //0.1f + fiber[index] * (1.0f - signal) / 2.0f;
		rhom[index] = d_basal_m * Gphi + sign2 * fiber2[index] * ( -sin(2.0f * M_PI * t / period2)) + sign1 * fiber1[index] *  ( -sin(2.0f * M_PI * t / period1)) + fiber3[index]; //0.1f + fiber[index] * (1.0f - signal) / 2.0f;

	}
}

__global__ void rhoa_osci(float *rhoa, float *rhom, float *phi, float x_sin, float x_cos, float t, float rhoa_period, float *x, float Lx, int m, int n){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j*m;
	if( i < m && j < n){
		rhoa[index] = 0.0f; 
		rhom[index] = 0.0f;
		if(phi[index] > 1e-4){
			float signal = cos(2*M_PI*t/rhoa_period);
			if(signal > 0.0f)
				rhoa[index] = (1.0f + signal) / 2.0f;
			//if(signal > 0.0f)
			//	rhoa[index] = tanh(signal / 0.2f);//(1.0f + signal) / 2.0f;
			
		
			//if(signal < -0.5f)
			float signal_m = sin(2*M_PI*t/rhoa_period);
			if(signal_m<= 0.0f)	
				rhom[index] = (1.0f - signal_m) / 2.0f;
			//rhoa[index] = tanh(signal / 0.2f);
			//if (signal > -0.5f)
			//	rhoa[index] = 1.0f;//(signal + 1.0f) / 2.0f;
		}
	}
}

__global__ void cylinder_phase(float *output, float start_x, float start_y, float end_x, float end_y, float radius, float *x, float *y, int Nx, int Ny){
	//a cynlinder-like fiber with radius r and starting point (x0,y0) ending point (xend,yend). (x0,y0) can be chosen to be the mass center of cell
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx ;

	if( i < Nx && j < Ny){
		float x1 = end_x - start_x;
		float y1 = end_y - start_y;

		float x2 = x[i] - start_x;
		float y2 = y[j] - start_y;

		float axia_len = sqrt(x1 * x1 + y1 * y1); //the axis length
		float rdis = sqrt((x1 * y2 - y1 * x2) * (x1 * y2 - y1 * x2));
		rdis = rdis / axia_len; //the radius-dir distance

		float alen = (x1 * x2 + y1 * y2) / axia_len; //the axial distance

		output[index] = (0.5f + 0.5f * tanh(3.0f * (radius - rdis) / d_epsilon)) * (0.5f + 0.5f * tanh(3.0f * (axia_len / 2.0f - abs(alen - axia_len / 2.0f)) / d_epsilon));
	}
}

__global__ void disc_phase(float *output, int center_idx, float radius, float *x, float *y, int Nx, int Ny){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx ;

	if( i < Nx && j < Ny){
		int x_idx = (center_idx - 1 ) % Nx;
		int y_idx = (center_idx - 1 ) / Nx;
		float dis = sqrt((x[i] - x[x_idx])  * (x[i] - x[x_idx]) + (y[j] - y[y_idx]) * (y[j] - y[y_idx]));
		output[index] = 0.5f + 0.5f * tanh(3.0f * (radius - dis) / d_epsilon);
	}
}

/*----------------------------------------------------------------------------------------------*/
/*Don't Change The Code After This Line if You Don't Want to Change the Differentiation Methods*/
/*----------------------------------------------------------------------------------------------*/

__global__ void set_unit(float *output, float alpha, int Nx, int Ny){
		int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx ;

	if( i < Nx && j < Ny){
		output[index] = alpha;
	}
}

__global__ void curv(float *curv, float *phi_gradx, float *phi_grady, float *absdphi, int Nx, int Ny, float dx, float dy){
	//from gradient obtained by fft
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j*Nx;
	//f: forward; b:backward
	int fdx = i + 1; if (i == Nx - 1) fdx = 0;
	fdx = fdx + j*Nx;
	int bdx = i - 1; if (i == 0) bdx = Nx - 1;
	bdx = bdx + j*Nx;
	int fdy = j + 1; if (j == Ny - 1) fdy = 0;
	fdy = i + fdy*Nx;
	int bdy = j - 1; if (j == 0) bdy = Ny - 1;
	bdy = i + bdy*Nx;

	curv[index] = 0.0f;
	if (absdphi[index] >= 0.01 && absdphi[fdx] >= 0.01 && absdphi[bdx] >= 0.01 && absdphi[fdy] >= 0.01 && absdphi[bdy] >= 0.01)
		curv[index] = -(phi_gradx[fdx] / absdphi[fdx] - phi_gradx[bdx] / absdphi[bdx]) / 2.0f / dx -
		(phi_grady[fdy] / absdphi[fdy] - phi_grady[bdy] / absdphi[bdy]) / 2.0f / dy;
	
}

__global__ void curv(float *curv, float *phi, float *absdphi, int m, int n, float dx, float dy){
	//from gradient obtained by fft
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j*m;
	if(i<m && j<n){
		curv[index] = 0.0f;
		if(absdphi[index]>0.001f){
			int r_dx = i + 1; if (i == m - 1) r_dx = 0;
			int l_dx = i - 1; if (i == 0) l_dx = m - 1;
			int u_dy = j + 1; if (j == n - 1) u_dy = 0;
			int d_dy = j - 1; if (j == 0) d_dy = n - 1;
			//geth the 9 points needed for curvature calculation
			float phi_i_j=phi[index];          //(i,j)
			float phi_ip1_j=phi[r_dx+j*m];      //(i+1,j)
			float phi_im1_j=phi[l_dx+j*m];      //(i-1,j)
			float phi_i_jp1=phi[i+u_dy*m];      //(i,j+1)
			float phi_i_jm1=phi[i+d_dy*m];      //(i,j-1)
			float phi_ip1_jp1=phi[r_dx+u_dy*m];  //(i+1,j+1)
			float phi_ip1_jm1=phi[r_dx+d_dy*m];  //(i+1,j-1)
			float phi_im1_jp1=phi[l_dx+u_dy*m];  //(i-1,j+1)
			float phi_im1_jm1=phi[l_dx+d_dy*m];  //(i-1,j-1)
			
			float phix_iphalf_j = (phi_ip1_j - phi_i_j  )/dx;
			float phix_imhalf_j = (phi_i_j   - phi_im1_j)/dx;
			float phiy_i_jphalf = (phi_i_jp1 - phi_i_j  )/dy;
			float phiy_i_jmhalf = (phi_i_j   - phi_i_jm1)/dy;

			float phiy_iphalf_j = (phi_ip1_jp1 + phi_i_jp1   - phi_ip1_jm1 - phi_i_jm1  )/(4.0f*dy);
			float phiy_imhalf_j = (phi_i_jp1   + phi_im1_jp1 - phi_i_jm1   - phi_im1_jm1)/(4.0f*dy);
			float phix_i_jphalf = (phi_ip1_jp1 + phi_ip1_j   - phi_im1_jp1 - phi_im1_j  )/(4.0f*dx);
			float phix_i_jmhalf = (phi_ip1_j   + phi_ip1_jm1 - phi_im1_j   - phi_im1_jm1)/(4.0f*dx);

			float grad_phi_abs_iphalf_j = sqrt( phix_iphalf_j * phix_iphalf_j + phiy_iphalf_j * phiy_iphalf_j );
			float grad_phi_abs_imhalf_j = sqrt( phix_imhalf_j * phix_imhalf_j + phiy_imhalf_j * phiy_imhalf_j);
			float grad_phi_abs_i_jphalf = sqrt( phix_i_jphalf * phix_i_jphalf + phiy_i_jphalf * phiy_i_jphalf );
			float grad_phi_abs_i_jmhalf = sqrt( phix_i_jmhalf * phix_i_jmhalf + phiy_i_jmhalf * phiy_i_jmhalf );

			curv[index] = - ( phix_iphalf_j / grad_phi_abs_iphalf_j - phix_imhalf_j / grad_phi_abs_imhalf_j )/dx - ( phiy_i_jphalf / grad_phi_abs_i_jphalf - phiy_i_jmhalf / grad_phi_abs_i_jmhalf )/dy;
		}
	}
}

__global__ void Gphi(float *output, float *phi, int Nx, int Ny){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx ;

	if( i < Nx && j < Ny){
		output[index] = 18.0f * phi[index] * phi[index] * (1.0f - phi[index]) * (1.0f - phi[index]) / d_epsilon;
	}
}

void fftcoeffs(float *d_d1x, float *d_d1y, float *d_d2x, float *d_d2y){
	//FFT coefficients, d_d1x, d_d1y are device 1-st derivatie grid matrixes, d_d2x, d_d2y are device 2-nd derivative matrix
	float *h_kx2 = (float *)malloc(Nx*sizeof(float));
	for (int i = 0; i <= Nx / 2; i++) h_kx2[i] = (float)i * M_PI / Lx;
	for (int i = Nx / 2 + 1; i < Nx; i++) h_kx2[i] = ((float)i - (float)Nx) * M_PI / Lx;
	hipMemcpy(d_d2x, h_kx2, Nx*sizeof(float), hipMemcpyHostToDevice);

	float *h_ky2 = (float *)malloc(Ny*sizeof(float));
	for (int i = 0; i <= Ny / 2; i++) h_ky2[i] = (float)i * M_PI / Ly;
	for (int i = Ny / 2 + 1; i < Ny; i++) h_ky2[i] = ((float)i - (float)Ny) * M_PI / Ly;
	hipMemcpy(d_d2y, h_ky2, Ny*sizeof(float), hipMemcpyHostToDevice);

	float *h_kx1 = (float *)malloc(Nx*sizeof(float));
	for (int i = 0; i < Nx / 2; i++) h_kx1[i] = (float)i * M_PI / Lx;
	h_kx1[Nx / 2] = 0.0f;
	for (int i = Nx / 2 + 1; i < Nx; i++) h_kx1[i] = ((float)i - (float)Nx) * M_PI / Lx;
	hipMemcpy(d_d1x, h_kx1, Nx*sizeof(float), hipMemcpyHostToDevice);

	float *h_ky1 = (float *)malloc(Ny*sizeof(float));
	for (int i = 0; i < Ny / 2; i++)h_ky1[i] = (float)i * M_PI / Ly;
	h_ky1[Ny / 2] = 0.0f;
	for (int i = Ny / 2 + 1; i < Ny; i++) h_ky1[i] = ((float)i - (float)Ny) * M_PI / Ly;
	hipMemcpy(d_d1y, h_ky1, Ny*sizeof(float), hipMemcpyHostToDevice);

	free(h_kx1);
	free(h_kx2);
	free(h_ky1);
	free(h_ky2);
}

__global__ void fftR2C_deriv1(hipfftComplex *input, hipfftComplex *output, int dim, float *coeff, int Nx, int Ny){
	//x:dim=1; y:dim=2;
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j*(Nx / 2 + 1);
	int dir[2]={i,j};
	if (i < (Nx / 2 + 1) && j < Ny){
		hipfftComplex temp = input[index];
		output[index].x = -temp.y*coeff[dir[dim-1]] / (float)Nx / (float)Ny;
		output[index].y = temp.x*coeff[dir[dim-1]] / (float)Nx / (float)Ny;
	}
}

__global__ void fftR2C_lap(hipfftComplex *input, hipfftComplex *output, float *coeff_x, float *coeff_y, int Nx, int Ny){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j*(Nx / 2 + 1);
	if (i < (Nx / 2 + 1) && j < Ny){
		hipfftComplex temp = input[index];
		float coeff = coeff_x[i] * coeff_x[i] + coeff_y[j] * coeff_y[j];
		output[index].x = -temp.x*coeff / (float)Nx / (float)Ny;
		output[index].y = -temp.y*coeff / (float)Nx / (float)Ny;
	}
}

void grad_fft(float *output_x, float *output_y, float *input, float *deriv_vec_x, float *deriv_vec_y, hipfftComplex *R2C_buffer, hipfftComplex *C2R_buffer, int Nx, int Ny){
	
	hipfftExecR2C(plan_R2C, input, R2C_buffer);

	fftR2C_deriv1 <<<blocks, threadsperblock >>>(R2C_buffer, C2R_buffer,1, deriv_vec_x, Nx, Ny);
	hipfftExecC2R(plan_C2R, C2R_buffer, output_x);

	fftR2C_deriv1 <<<blocks, threadsperblock >>>(R2C_buffer, C2R_buffer,2, deriv_vec_y, Nx, Ny);
	hipfftExecC2R(plan_C2R, C2R_buffer, output_y);
}

void lap_fft(float *output, float *input, float *deriv_vec_x, float *deriv_vec_y, hipfftComplex *R2C_buffer, hipfftComplex *C2R_buffer, int Nx, int Ny){
	hipfftExecR2C(plan_R2C, input, R2C_buffer);
	fftR2C_lap <<<blocks, threadsperblock >>>(R2C_buffer, C2R_buffer, deriv_vec_x,deriv_vec_y, Nx, Ny);
	hipfftExecC2R(plan_C2R, C2R_buffer, output);
}

void div_fft(float *output, float *input_x, float *input_y, float *deriv_vec_x, float *deriv_vec_y, hipfftComplex *R2C_buffer, hipfftComplex *C2R_buffer, int Nx, int Ny){
	//in-space transform, notice that input_x and input_y will change values when function is called
	hipfftExecR2C(plan_R2C, input_x, R2C_buffer);
	fftR2C_deriv1 <<<blocks, threadsperblock >>>(R2C_buffer, C2R_buffer, 1, deriv_vec_x, Nx, Ny);
	hipfftExecC2R(plan_C2R, C2R_buffer, input_x);

	hipfftExecR2C(plan_R2C, input_y, R2C_buffer);
	fftR2C_deriv1 <<<blocks, threadsperblock >>>(R2C_buffer, C2R_buffer, 2, deriv_vec_y, Nx, Ny);
	hipfftExecC2R(plan_C2R, C2R_buffer, input_y);

	add2matrix<<<blocks, threadsperblock>>>(output, input_x, input_y);
}

void div_pst_fft(float *output, float *c_pst, float *rho, float coeff_diff, float *deriv_vec_x2, float *deriv_vec_y2, float *buffer_x, float *buffer_y, hipfftComplex *R2C_buffer, hipfftComplex *C2R_buffer, int Nx, int Ny){
	//notice this differential process uses the 2-nd kind of derivative vector, corresponding to 2-nd order derivative
	grad_fft(buffer_x, buffer_y, rho, deriv_vec_x2, deriv_vec_y2, R2C_buffer, C2R_buffer, Nx, Ny);

	matrix_product<<<blocks, threadsperblock >>>(buffer_x, buffer_x, c_pst, coeff_diff, Nx, Ny);
	matrix_product<<<blocks, threadsperblock >>>(buffer_y, buffer_y, c_pst, coeff_diff, Nx, Ny);

	div_fft(output, buffer_x, buffer_y, deriv_vec_x2, deriv_vec_y2, R2C_buffer, C2R_buffer, Nx, Ny);
}

__global__ void div_pst_fd(float *output, float *c_pst, float *rho, float coeff_diff, float dx, float dy, int Nx, int Ny){
	//finite differential of the diffusion term
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j * Nx;

	int r_dx = i + 1; if (i == Nx - 1) r_dx = 0;
	r_dx = r_dx + j * Nx;
	int l_dx = i - 1; if (i == 0) l_dx = Nx - 1;
	l_dx = l_dx + j * Nx;
	int u_dy = j + 1; if (j == Ny - 1) u_dy = 0;
	u_dy = i + u_dy * Nx;
	int d_dy = j - 1; if (j == 0) d_dy = Ny - 1;
	d_dy = i + d_dy * Nx;

	output[index] = coeff_diff * ((c_pst[r_dx] + c_pst[index]) / 2.0f * (rho[r_dx] - rho[index]) / dx - (c_pst[l_dx] + c_pst[index]) / 2.0f * (rho[index] - rho[l_dx]) / dx) / dx +
		coeff_diff * ((c_pst[u_dy] + c_pst[index]) / 2.0f * (rho[u_dy] - rho[index]) / dy - (c_pst[d_dy] + c_pst[index]) / 2.0f * (rho[index] - rho[d_dy]) / dy) / dy;

}

void par_deriv1_fft(float *output, float *input, int dim, float *deriv_vec, hipfftComplex *R2C_buffer, hipfftComplex *C2R_buffer, int Nx, int Ny){
	
	hipfftExecR2C(plan_R2C, input, R2C_buffer);
	if(dim == 1){
		fftR2C_deriv1 <<<blocks, threadsperblock >>>(R2C_buffer, C2R_buffer,1, deriv_vec, Nx, Ny);
		hipfftExecC2R(plan_C2R, C2R_buffer, output);
	}
   	 if(dim == 2){
	    fftR2C_deriv1<<<blocks, threadsperblock >>>(R2C_buffer, C2R_buffer,2, deriv_vec, Nx, Ny);
		hipfftExecC2R(plan_C2R, C2R_buffer, output);
	}
}

__global__ void poissonR2C(hipfftComplex *input, hipfftComplex *output, float *coeff_x, float *coeff_y, float coeff0, float coeff2, int Nx, int Ny){
    //equation of coeff0*u - coeff2*Delta(u) = f
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = i + j*(Nx / 2 + 1);
	if (i < (Nx / 2 + 1) && j < Ny){
		float coeff = coeff_x[i] * coeff_x[i] + coeff_y[j] * coeff_y[j];
		hipfftComplex temp = input[index];
		output[index].x = temp.x / (coeff0 + coeff2 * coeff) / (float)Nx / (float)Ny;
		output[index].y = temp.y / (coeff0 + coeff2 * coeff) / (float)Nx / (float)Ny;
	}
}

void poisson_sol(float *sol, float *rhs, float coeff0, float coeff2, float *coeff_x, float *coeff_y, hipfftComplex *R2C_buffer, hipfftComplex *C2R_buffer, int Nx, int Ny){
	hipfftExecR2C(plan_R2C, rhs, R2C_buffer);
	poissonR2C <<<blocks, threadsperblock >>>(R2C_buffer, C2R_buffer, coeff_x, coeff_y, coeff0, coeff2, Nx, Ny);
	hipfftExecC2R(plan_C2R, C2R_buffer, sol);
}

/*----------------------------------------------------------------------------------------------*/
/*----------------------------------------------------------------------------------------------*/
